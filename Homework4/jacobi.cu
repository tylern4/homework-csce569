#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <sys/timeb.h>
#define REAL float
#define BLOCK_SIZE 16

/* compile the program using the following command
 *    nvcc jacobi.cu -lpthread -o jacobi
*/

/* read timer in second */
double read_timer() {
  struct timeb tm;
  ftime(&tm);
  return (double)tm.time + (double)tm.millitm / 1000.0;
}

/* read timer in ms */
double read_timer_ms() {
  struct timeb tm;
  ftime(&tm);
  return (double)tm.time * 1000.0 + (double)tm.millitm;
}

/************************************************************
 * program to solve a finite difference
 * discretization of Helmholtz equation :
 * (d2/dx2)u + (d2/dy2)u - alpha u = f
 * using Jacobi iterative method.
 *
 * Modified: Sanjiv Shah,       Kuck and Associates, Inc. (KAI), 1998
 * Author:   Joseph Robicheaux, Kuck and Associates, Inc. (KAI), 1998
 *
 * This c version program is translated by
 * Chunhua Liao, University of Houston, Jan, 2005
 *
 * Directives are used in this code to achieve parallelism.
 * All do loops are parallelized with default 'static' scheduling.
 *
 * Input :  n - grid dimension in x direction
 *          m - grid dimension in y direction
 *          alpha - Helmholtz constant (always greater than 0.0)
 *          tol   - error tolerance for iterative solver
 *          relax - Successice over relaxation parameter
 *          mits  - Maximum iterations for iterative solver
 *
 * On output
 *       : u(n,m) - Dependent variable (solutions)
 *       : f(n,m) - Right hand side function
 *************************************************************/

// flexible between REAL and double
#define DEFAULT_DIMSIZE 256

void print_array(char *title, char *name, REAL *A, long n, long m) {
  printf("%s:\n", title);
  long i, j;
  for (i = 0; i < n; i++) {
    for (j = 0; j < m; j++) {
      printf("%s[%ld][%ld]:%f  ", name, i, j, A[i * m + j]);
    }
    printf("\n");
  }
  printf("\n");
}

/*      subroutine initialize (n,m,alpha,dx,dy,u,f)
 ******************************************************
 * Initializes data
 * Assumes exact solution is u(x,y) = (1-x^2)*(1-y^2)
 *
 ******************************************************/
void initialize(long n, long m, REAL alpha, REAL *dx, REAL *dy, REAL *u_p, REAL *f_p) {
  long i;
  long j;
  long xx;
  long yy;
  REAL(*u)[m] = (REAL(*)[m])u_p;
  REAL(*f)[m] = (REAL(*)[m])f_p;

  // double PI=3.1415926;
  *dx = (2.0 / (n - 1));
  *dy = (2.0 / (m - 1));
  /* Initialize initial condition and RHS */
  //#pragma omp parallel for private(xx,yy,j,i)
  for (i = 0; i < n; i++)
    for (j = 0; j < m; j++) {
      xx = ((int)(-1.0 + (*dx * (i - 1))));
      yy = ((int)(-1.0 + (*dy * (j - 1))));
      u[i][j] = 0.0;
      f[i][j] = (((((-1.0 * alpha) * (1.0 - (xx * xx))) * (1.0 - (yy * yy))) - (2.0 * (1.0 - (xx * xx)))) -
                 (2.0 * (1.0 - (yy * yy))));
    }
}

/*  subroutine error_check (n,m,alpha,dx,dy,u,f)
 implicit none
 ************************************************************
 * Checks error between numerical and exact solution
 *
 ************************************************************/
double error_check(long n, long m, REAL alpha, REAL dx, REAL dy, REAL *u_p, REAL *f_p) {
  int i;
  int j;
  REAL xx;
  REAL yy;
  REAL temp;
  double error;
  error = 0.0;
  REAL(*u)[m] = (REAL(*)[m])u_p;
// REAL(*f)[m] = (REAL(*)[m])f_p;
#pragma omp parallel for private(xx, yy, temp, j, i) reduction(+ : error)
  for (i = 0; i < n; i++)
    for (j = 0; j < m; j++) {
      xx = (-1.0 + (dx * (i - 1)));
      yy = (-1.0 + (dy * (j - 1)));
      temp = (u[i][j] - ((1.0 - (xx * xx)) * (1.0 - (yy * yy))));
      error = (error + (temp * temp));
    }
  error = (sqrt(error) / (n * m));
  return error;
}
void jacobi_seq(long n, long m, REAL dx, REAL dy, REAL alpha, REAL relax, REAL *u_p, REAL *f_p, REAL tol, int mits);
void jacobi_cuda(long n, long m, REAL dx, REAL dy, REAL alpha, REAL relax, REAL *u_p, REAL *f_p, REAL tol, int mits);

int main(int argc, char *argv[]) {
  long n = DEFAULT_DIMSIZE;
  long m = DEFAULT_DIMSIZE;
  REAL alpha = 0.0543;
  REAL tol = 0.0000000001;
  REAL relax = 1.0;
  int mits = 5000;

  if (argc == 2) {
    sscanf(argv[1], "%ld", &n);
    m = n;
  } else if (argc == 3) {
    sscanf(argv[1], "%ld", &n);
    sscanf(argv[2], "%ld", &m);
  } else if (argc == 4) {
    sscanf(argv[1], "%ld", &n);
    sscanf(argv[2], "%ld", &m);
    sscanf(argv[3], "%g", &alpha);
  } else if (argc == 5) {
    sscanf(argv[1], "%ld", &n);
    sscanf(argv[2], "%ld", &m);
    sscanf(argv[3], "%g", &alpha);
    sscanf(argv[4], "%g", &tol);
  } else if (argc == 6) {
    sscanf(argv[1], "%ld", &n);
    sscanf(argv[2], "%ld", &m);
    sscanf(argv[3], "%g", &alpha);
    sscanf(argv[4], "%g", &tol);
    sscanf(argv[5], "%g", &relax);
  } else if (argc == 7) {
    sscanf(argv[1], "%ld", &n);
    sscanf(argv[2], "%ld", &m);
    sscanf(argv[3], "%g", &alpha);
    sscanf(argv[4], "%g", &tol);
    sscanf(argv[5], "%g", &relax);
    sscanf(argv[6], "%d", &mits);
  } else {
    fprintf(stderr, "Usage: jacobi [<n> <m> <alpha> <tol> <relax> <mits>]\n");
    fprintf(stderr, "\tn - grid dimension in x direction, default: %ld\n", n);
    fprintf(stderr, "\tm - grid dimension in y direction, default: n if provided or %ld\n", m);
    fprintf(stderr, "\talpha - Helmholtz constant (always greater than 0.0), default: %g\n", alpha);
    fprintf(stderr, "\ttol   - error tolerance for iterative solver, default: %g\n", tol);
    fprintf(stderr, "\trelax - Successice over relaxation parameter, default: %g\n", relax);
    fprintf(stderr, "\tmits  - Maximum iterations for iterative solver, default: %d\n", mits);
  }

  printf("jacobi %ld %ld %g %g %g %d\n", n, m, alpha, tol, relax, mits);
  printf("---------------------------------------------------------------\n");
  /** init the array */

  REAL *u = (REAL *)malloc(sizeof(REAL) * n * m);
  REAL *f = (REAL *)malloc(sizeof(REAL) * n * m);

  REAL *ucuda = (REAL *)malloc(sizeof(REAL) * n * m);
  REAL *fcuda = (REAL *)malloc(sizeof(REAL) * n * m);

  REAL dx; /* grid spacing in x direction */
  REAL dy; /* grid spacing in y direction */

  initialize(n, m, alpha, &dx, &dy, u, f);

  memcpy(ucuda, u, n * m * sizeof(REAL));
  memcpy(fcuda, f, n * m * sizeof(REAL));

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  // Free speed
  REAL *cuda_temp = (REAL *)malloc(sizeof(REAL));
  hipMalloc(&cuda_temp, (sizeof(REAL)));

  printf("===================== Sequential Execution =====================\n");
  double elapsed_seq = read_timer_ms();
  jacobi_seq(n, m, dx, dy, alpha, relax, u, f, tol, mits);
  elapsed_seq = read_timer_ms() - elapsed_seq;
  printf("\n");

  printf("===================== GPU CUDA Execution =====================\n");

  double elapsed_cuda = read_timer_ms();
  jacobi_cuda(n, m, dx, dy, alpha, relax, ucuda, fcuda, tol, mits);
  elapsed_cuda = read_timer_ms() - elapsed_cuda;
  printf("\n");

#if CORRECTNESS_CHECK
  print_array("Sequential Run", "u", (REAL *)u, n, m);
  print_array("GPU Run       ", "ucuda", (REAL *)ucuda, n, m);
#endif

  double flops = mits * (n - 2) * (m - 2) * 13;
  printf("---------------------------------------------------------------\n");
  printf("Performance:\tRuntime(ms)\tMFLOPS\t\tError\n");
  printf("---------------------------------------------------------------\n");
  printf("base:\t\t%.2f\t\t%.2f\t\t%g\n", elapsed_seq, flops / (1.0e3 * elapsed_seq),
         error_check(n, m, alpha, dx, dy, u, f));
  printf("GPU :\t\t%.2f\t\t%.2f\t\t%g\n", elapsed_cuda, flops / (1.0e3 * elapsed_cuda),
         error_check(n, m, alpha, dx, dy, ucuda, fcuda));

  free(u);
  free(f);
  free(ucuda);
  free(fcuda);
  hipFree(cuda_temp);

  return 0;
}

/*      subroutine jacobi (n,m,dx,dy,alpha,omega,u,f,tol,mits)
 ******************************************************************
 * Subroutine HelmholtzJ
 * Solves poisson equation on rectangular grid assuming :
 * (1) Uniform discretization in each direction, and
 * (2) Dirichlect boundary conditions
 *
 * Jacobi method is used in this routine
 *
 * Input : n,m   Number of grid points in the X/Y directions
 *         dx,dy Grid spacing in the X/Y directions
 *         alpha Helmholtz eqn. coefficient
 *         omega Relaxation factor
 *         f(n,m) Right hand side function
 *         u(n,m) Dependent variable/Solution
 *         tol    Tolerance for iterative solver
 *         mits  Maximum number of iterations
 *
 * Output : u(n,m) - Solution
 *****************************************************************/
void jacobi_seq(long n, long m, REAL dx, REAL dy, REAL alpha, REAL omega, REAL *u_p, REAL *f_p, REAL tol, int mits) {
  long i, j, k;
  REAL error;
  REAL ax;
  REAL ay;
  REAL b;
  REAL resid;
  REAL uold[n][m];
  REAL(*u)[m] = (REAL(*)[m])u_p;
  REAL(*f)[m] = (REAL(*)[m])f_p;
  /*
   * Initialize coefficients */
  /* X-direction coef */
  ax = (1.0 / (dx * dx));
  /* Y-direction coef */
  ay = (1.0 / (dy * dy));
  /* Central coeff */
  b = (((-2.0 / (dx * dx)) - (2.0 / (dy * dy))) - alpha);
  error = (10.0 * tol);
  k = 1;
  while ((k <= mits) && (error > tol)) {
    error = 0.0;

    /* Copy new solution into old */
    for (i = 0; i < n; i++)
      for (j = 0; j < m; j++) uold[i][j] = u[i][j];

    for (i = 1; i < (n - 1); i++)
      for (j = 1; j < (m - 1); j++) {
        resid =
            (ax * (uold[i - 1][j] + uold[i + 1][j]) + ay * (uold[i][j - 1] + uold[i][j + 1]) + b * uold[i][j] - f[i][j]) / b;
        // printf("i: %d, j: %d, resid: %f\n", i, j, resid);

        u[i][j] = uold[i][j] - omega * resid;
        error = error + resid * resid;
      }
    /* Error check */
    if (k % 500 == 0) printf("Finished %ld iteration with error: %g\n", k, error);
    error = sqrt(error) / (n * m);

    k = k + 1;
  } /*  End iteration loop */
  printf("Total Number of Iterations: %ld\n", k);
  printf("Residual: %.15g\n", error);
}

/**
 * TODO #1: jacobi_kernel implementation of the double-nested loop for
 * computation
 */
//__constant__ REAL *cuda_f;
__global__ void jacobi_kernel(long n, long m, REAL ax, REAL ay, REAL b, REAL omega, REAL *u, REAL *uold, REAL *resid,
                              REAL *cuda_f) {
  long i, j;
  i = blockIdx.x * blockDim.x + threadIdx.x;
  j = blockIdx.y * blockDim.y + threadIdx.y;

  if (i == 0 || j == 0) return;
  if (i >= (n - 1) || j >= (m - 1)) return;

  // if (i > 0 && i < (n - 1)) {
  //  if (i > 0 && j < (m - 1)) {
  resid[i * n + j] = (ax * (uold[(i - 1) * n + j] + uold[(i + 1) * n + j]) +
                      ay * (uold[i * n + (j - 1)] + uold[i * n + (j + 1)]) + b * uold[i * n + j] - cuda_f[i * n + j]) /
                     b;
  u[i * n + j] = uold[i * n + j] - omega * resid[i * n + j];
  //  }
  //}
}
void jacobi_cuda(long n, long m, REAL dx, REAL dy, REAL alpha, REAL omega, REAL *u_p, REAL *f_p, REAL tol, int mits) {
  long i, j, k;
  REAL error;
  REAL ax;
  REAL ay;
  REAL b;
  REAL *resid = (REAL *)malloc((sizeof(REAL) * n * m));
  REAL *uold = (REAL *)malloc((sizeof(REAL) * n * m));
  REAL *temp;
  REAL(*u)[m] = (REAL(*)[m])u_p;
  REAL(*f)[m] = (REAL(*)[m])f_p;

  /*
   * Initialize coefficients */
  /* X-direction coef */
  ax = (1.0 / (dx * dx));
  /* Y-direction coef */
  ay = (1.0 / (dy * dy));
  /* Central coeff */
  b = (((-2.0 / (dx * dx)) - (2.0 / (dy * dy))) - alpha);
  error = (10.0 * tol);
  k = 1;
  /* TODO #2: CUDA memory allocation for u, f and uold and copy data for u and f
   * from host memory to GPU memory, depending on how error
   * will be calculated (see below), a [n][m] array or a one-element array need
   * to be allocated as well. */
  int size = (sizeof(REAL) * n * m);
  REAL *cuda_u;
  REAL *cuda_f;
  REAL *cuda_uold;
  REAL *cuda_resid;

  // Copy u to cuda memory
  hipMalloc((void **)&cuda_u, size);
  hipMemcpy(cuda_u, u, size, hipMemcpyHostToDevice);

  hipMalloc((void **)&cuda_f, size);
  hipMemcpy(cuda_f, f, size, hipMemcpyHostToDevice);

  hipMalloc((void **)&cuda_uold, size);
  hipMalloc((void **)&cuda_resid, size);

  /* TODO #4: set 16x16 threads/block and n/16 x m/16 blocks/grid for GPU
   * computation (assuming n and m are dividable by 16 */
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid(n / dimBlock.x, m / dimBlock.y);

  while ((k <= mits) && (error > tol)) {
    error = 0.0;

    /* TODO #3: swap u and uold */
    temp = cuda_u;
    cuda_u = cuda_uold;
    cuda_uold = temp;
    /* TODO #5: launch jacobi_kernel */
    jacobi_kernel << <dimGrid, dimBlock>>> (n, m, ax, ay, b, omega, cuda_u, cuda_uold, cuda_resid, cuda_f);
    /* TODO #6: compute error on CPU or GPU. error is calculated by accumulating
    *          resid*resid computed by each thread. There are multiple
    * approaches to compute the error. E.g. 1). A array of resid[n][m]
    *          could be allocated and store the resid computed by each thread.
    * After the computation, all the resids in the array are
    *          accumulated on either CPU or GPU. 2). A simpler implementation
    * could be just using CUDA atomicAdd, check.
    *
 (http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#atomic-functions
    */
    /* Error check */
    /* TODO #7: copy the error from GPU to CPU */
    hipMemcpy(resid, cuda_resid, size, hipMemcpyDeviceToHost);
    for (i = 1; i < (n - 1); i++)
      for (j = 1; j < (m - 1); j++) {
        error = error + resid[i * n + j] * resid[i * n + j];
      }

    if (k % 500 == 0) printf("Finished %ld iteration with error: %g\n", k, error);
    error = sqrt(error) / (n * m);
    k = k + 1;
  } /*  End iteration loop */
    /* TODO #8: GPU memory deallocation */
  hipFree(cuda_u);
  hipFree(cuda_f);
  hipFree(cuda_uold);
  hipFree(cuda_resid);
  printf("Total Number of Iterations: %ld\n", k);
  printf("Residual: %.15g\n", error);
}
