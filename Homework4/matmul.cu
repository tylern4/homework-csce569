#include "hip/hip_runtime.h"
/*
 * Rectangular matrix multiplication
 * A[M][K] * B[k][N] = C[M][N]
 *
 */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/timeb.h>
#include <string.h>

/* read timer in second */
double read_timer() {
    struct timeb tm;
    ftime(&tm);
    return (double) tm.time + (double) tm.millitm / 1000.0;
}

/* read timer in ms */
double read_timer_ms() {
    struct timeb tm;
    ftime(&tm);
    return (double) tm.time * 1000.0 + (double) tm.millitm;
}

#define REAL float

void init(int M, int N, REAL * A) {
    int i, j;

    for (i = 0; i < M; i++) {
        for (j = 0; j < N; j++) {
            A[i*N+j] = (REAL) drand48();
        }
    }
}

double maxerror(int M, int N, REAL * A, REAL *B) {
    int i, j;
    double error = 0.0;

    for (i = 0; i < M; i++) {
        for (j = 0; j < N; j++) {
            double diff = (A[i*N+j] - B[i*N+j]) / A[i*N+j];
            if (diff < 0)
                diff = -diff;
            if (diff > error)
                error = diff;
        }
    }
    return error;
}

void matmul_base(int N, REAL *A, REAL * B, REAL *C);
void matmul_openmp(int N, REAL *A, REAL *B, REAL *C, int num_tasks);
void matmul_cuda_v1_vanilla(int N, REAL *A, REAL *B, REAL *C);
void matmul_cuda_v1_shmem(int N, REAL *A, REAL *B, REAL *C);
void matmul_cuda_v1_cublas(int N, REAL *A, REAL *B, REAL *C);

int main(int argc, char *argv[]) {
    int N;
    int num_tasks = 5; /* 5 is default number of tasks */
    double elapsed_base, elapsed_openmp;
    //double elapsed_cuda_v1, elapsed_cuda_v2, elapsed_cuda_v3; /* for timing */
    if (argc < 2) {
        fprintf(stderr, "Usage: matmul <n> [<#tasks(%d)>]\n", num_tasks);
        exit(1);
    }
    N = atoi(argv[1]);
    if (argc > 2) num_tasks = atoi(argv[2]);
    REAL * heap_buffer = (REAL*)malloc(sizeof(REAL)*N*N*4); /* we use 5 matrix in this example */
    /* below is a cast from memory buffer to a 2-d row-major array */
    REAL *A = heap_buffer;
    REAL *B = &heap_buffer[N*N];
    REAL *C_base = &heap_buffer[2*N*N];
    REAL *C_openmp = &heap_buffer[3*N*N];

    srand48((1 << 12));
    init(N, N, A);
    init(N, N, B);

    /* example run */
    elapsed_base = read_timer();
    matmul_base(N, A, B, C_base);
    elapsed_base = (read_timer() - elapsed_base);

    elapsed_openmp = read_timer();
    matmul_openmp(N, A, B, C_openmp, num_tasks);
    elapsed_openmp = (read_timer() - elapsed_openmp);

    /* call and timing for the three CUDA versions */
    //TODO: call and time for matmul_cuda_v1_vanilla(int N, REAL *A, REAL *B, REAL *C);

    //TODO: call and time for matmul_cuda_v1_shmem(int N, REAL *A, REAL *B, REAL *C);

    //TODO: call and time for matmul_cuda_v1_cublas(int N, REAL *A, REAL *B, REAL *C);

    printf("======================================================================================================\n");
    printf("Matrix Multiplication: A[M][K] * B[k][N] = C[M][N], M=K=N=%d, %d threads/tasks\n", N, num_tasks);
    printf("------------------------------------------------------------------------------------------------------\n");
    printf("Performance:\t\tRuntime (ms)\t MFLOPS \t\tError (compared to base)\n");
    printf("------------------------------------------------------------------------------------------------------\n");
    printf("matmul_base:\t\t%4f\t%4f \t\t%g\n", elapsed_base * 1.0e3, ((((2.0 * N) * N) * N) / (1.0e6 * elapsed_base)), maxerror(N, N, C_base, C_base));
    printf("matmul_openmp:\t\t%4f\t%4f \t\t%g\n", elapsed_openmp * 1.0e3, ((((2.0 * N) * N) * N) / (1.0e6 * elapsed_openmp)), maxerror(N, N, C_base, C_openmp));
    /* TODO: put other printf statements for outputing results for GPU execution */
    free(heap_buffer);
    return 0;
}

void matmul_base(int N, REAL *A, REAL * B, REAL *C) {
    int i, j, k;
    for (i = 0; i < N; i++) {
        for (j = 0; j < N; j++) {
            REAL temp = 0.0;
            for (k = 0; k < N; k++) {
                temp += A[i*N+k] * B[k*N+j];
            }
            C[i*N+j] = temp;
        }
    }
}

void matmul_openmp(int N, REAL *A, REAL *B, REAL *C, int num_tasks) {
    int i, j, k;
#pragma omp parallel for shared(N,A,B,C,num_tasks) private(i,j,k) num_threads(num_tasks)
    for (i = 0; i < N; i++) {
        for (j = 0; j < N; j++) {
            REAL temp = 0.0;
            for (k = 0; k < N; k++) {
                temp += A[i*N+k] * B[k*N+j];
            }
            C[i*N+j] = temp;
        }
    }
}

/** 
  * TODO: kernel implementation 
  */
__global__ matmul_cuda_v1_vanilla_kernel( ... ) {

}
/*
 * call to kernel that uses GPU global memory
 */
void matmul_cuda_v1_vanilla(int N, REAL *A, REAL *B, REAL *C) {

}

/** 
  * TODO: kernel implementation 
  */
__global__ matmul_cuda_v2_shmem_kernel( ... ) {

}
/*
 * call to kernel that use GPU shared memory
 */
void matmul_cuda_v2_shmem(int N, REAL *A, REAL *B, REAL *C) {

}

/*
 * call to sgemm of cublas library 
 */
void matmul_cuda_v3_cublas(int N, REAL *A, REAL *B, REAL *C) {

}
